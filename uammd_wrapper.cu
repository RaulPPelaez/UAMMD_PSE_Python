/*Raul P. Pelaez 2021. This code exposes in a class the UAMMD's PSE module.
  Allows to compute the hydrodynamic displacements of a group of particles due to thermal fluctuations and/or forces acting on them.
  See example.cpp for usage instructions.
  See example.py for usage from python.
 */
#include <uammd.cuh>
#include <Integrator/BDHI/BDHI_PSE.cuh>
#include"uammd_interface.h"
using namespace uammd;
using PSE = BDHI::PSE;
using Parameters = PSE::Parameters;

struct Real3ToReal4{
  __host__ __device__ real4 operator()(real3 i){
    auto pr4 = make_real4(i);
    return pr4;
  }
};


Parameters toPSEParameters(PyParameters par){
  Parameters psepar;
  psepar.temperature = par.temperature;
  psepar.viscosity = par.viscosity;
  psepar.hydrodynamicRadius = par.hydrodynamicRadius;
  psepar.dt = par.dt;
  psepar.box = Box(make_real3(par.Lx, par.Ly, par.Lz));
  psepar.tolerance = par.tolerance;
  psepar.psi = par.psi;
  psepar.shearStrain = par.shearStrain;
  return psepar;  
}

struct UAMMD_PSE {
  using real = real;
  std::shared_ptr<System> sys;
  std::shared_ptr<ParticleData> pd;
  std::shared_ptr<PSE> pse;
  thrust::device_vector<real> d_MF;
  thrust::device_vector<real3> tmp;
  int numberParticles;
  hipStream_t st;
  UAMMD_PSE(PyParameters par, int numberParticles): numberParticles(numberParticles){
    this->sys = std::make_shared<System>();
    this->pd = std::make_shared<ParticleData>(numberParticles, sys);
    auto pg = std::make_shared<ParticleGroup>(pd, sys, "All");
    this->pse = std::make_shared<PSE>(pd, pg, sys, toPSEParameters(par));
    d_MF.resize(3*numberParticles);
    tmp.resize(numberParticles);
    CudaSafeCall(hipStreamCreate(&st));
  }

  void Mdot(const real* h_pos,
	    const real* h_F,
	    real* h_MF){
    uploadPosAndForceToUAMMD(h_pos, h_F);
    auto d_MF_ptr = (real3*)(thrust::raw_pointer_cast(d_MF.data()));
    pse->computeMF(d_MF_ptr, st);
    thrust::copy(d_MF.begin(), d_MF.end(), h_MF);
  }

  void MdotNearField(const real* h_pos,
		     const real* h_F,
		     real* h_MF){
    uploadPosAndForceToUAMMD(h_pos, h_F);
    auto d_MF_ptr = (real3*)(thrust::raw_pointer_cast(d_MF.data()));
    pse->computeMFNearField(d_MF_ptr, st);
    thrust::copy(d_MF.begin(), d_MF.end(), h_MF);
  }

  void MdotFarField(const real* h_pos,
		    const real* h_F,
		    real* h_MF){
    uploadPosAndForceToUAMMD(h_pos, h_F);
    auto d_MF_ptr = (real3*)(thrust::raw_pointer_cast(d_MF.data()));
    pse->computeMFFarField(d_MF_ptr, st);
    thrust::copy(d_MF.begin(), d_MF.end(), h_MF);
  }

  ~UAMMD_PSE(){
    hipDeviceSynchronize();
    hipStreamDestroy(st);
  }

private:
  void uploadPosAndForceToUAMMD(const real* h_pos, const real* h_F){
    auto pos = pd->getPos(access::location::gpu, access::mode::write);
    thrust::copy((real3*)h_pos, (real3*)h_pos + numberParticles, tmp.begin());
    thrust::transform(thrust::cuda::par.on(st), tmp.begin(), tmp.end(), pos.begin(), Real3ToReal4());
    if(h_F){
      auto forces = pd->getForce(access::location::gpu, access::mode::write);
      thrust::copy((real3*)h_F, (real3*)h_F + numberParticles, tmp.begin());
      thrust::transform(thrust::cuda::par.on(st), tmp.begin(), tmp.end(), forces.begin(), Real3ToReal4());
    }   
  }
};


UAMMD_PSE_Glue::UAMMD_PSE_Glue(PyParameters pypar, int numberParticles){
  pse = std::make_shared<UAMMD_PSE>(pypar, numberParticles);
}

void UAMMD_PSE_Glue::Mdot(const real* h_pos, const real* h_F, real* h_MF){
  pse->Mdot(h_pos, h_F, h_MF);
}

void UAMMD_PSE_Glue::MdotNearField(const real* h_pos, const real* h_F, real* h_MF){
  pse->MdotNearField(h_pos, h_F, h_MF);
}

void UAMMD_PSE_Glue::MdotFarField(const real* h_pos, const real* h_F, real* h_MF){
  pse->MdotFarField(h_pos, h_F, h_MF);
}
